#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/instance_norm_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


	template <typename Dtype>
	void InstanceNormLayer<Dtype>::Forward_gpu(
		const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* mutable_top_data = top[0]->mutable_gpu_data();
		const Dtype* top_data = top[0]->gpu_data();
		// define by GS
		const Dtype* weight_sum = SumMatrix_.gpu_data();

		Dtype* mutable_mu = Mu_.mutable_gpu_data();
		const Dtype* mu = Mu_.gpu_data();
		Dtype* mutable_expand_mu = Expand_Mu_.mutable_gpu_data();
		const Dtype* expand_mu = Expand_Mu_.gpu_data();

		Dtype* mutable_sigma = Sigma_.mutable_gpu_data();
		const Dtype* sigma = Sigma_.gpu_data();
		Dtype* mutable_expand_sigma = Expand_Sigma_.mutable_gpu_data();
		const Dtype* expand_sigma = Expand_Sigma_.gpu_data();

		Dtype* mutable_sigma2 = Sigma2_.mutable_gpu_data();
		const Dtype* sigma2 = Sigma2_.gpu_data();
		Dtype* mutable_expand_sigma2 = Expand_Sigma2_.mutable_gpu_data();
		const Dtype* expand_sigma2 = Expand_Sigma2_.gpu_data();

		Dtype* mutable_x_minus_mu = XMinusMu_.mutable_gpu_data();
		const Dtype* x_minus_mu = XMinusMu_.gpu_data();

		// top = bottom
		//if (bottom[0] != top[0]) {
		caffe_copy(bottom[0]->count(), bottom[0]->gpu_data(), top[0]->mutable_gpu_data());
		//}

		// calculate mu 
		caffe_gpu_gemm(CblasNoTrans, CblasTrans, 1, ch_*num_, imgSize_, Dtype(1. / imgSize_), SumMatrix_.gpu_data(), bottom[0]->gpu_data(), Dtype(0), Mu_.mutable_gpu_data());
		// expand mu from num_ * ch_ to num_ * ch_ * w_ * h_     // y = x - mu(x)
		caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, ch_*num_, imgSize_, 1, Dtype(-1), Mu_.gpu_data(), SumMatrix_.gpu_data(), Dtype(1), top[0]->mutable_gpu_data());
		// calculate sigma, use expand_sigma2 as a buffer to restore 
		caffe_gpu_powx(count_, top[0]->gpu_data(), Dtype(2), mutable_expand_sigma2);
		// calculate sum of variance in the range of spatial
		caffe_gpu_gemm(CblasNoTrans, CblasTrans, 1, ch_*num_, imgSize_, (Dtype)(1. / imgSize_), SumMatrix_.gpu_data(), Expand_Sigma2_.gpu_data(), (Dtype)0, Sigma2_.mutable_gpu_data());
		// add epsilon to the var
		caffe_gpu_add_scalar(ch_*num_, Dtype(epsilon_), Sigma2_.mutable_gpu_data());
		caffe_gpu_powx(ch_*num_, Sigma2_.gpu_data(), Dtype(0.5), Sigma_.mutable_gpu_data());
		// expand sigma from num_ * ch_ * 1 * 1 to num_ * ch_ * h_ * w_
		caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, ch_*num_, imgSize_, 1, (Dtype)1, Sigma_.gpu_data(), SumMatrix_.gpu_data(), (Dtype)0, Expand_Sigma_.mutable_gpu_data());
		// y = (x - mu(x)) / (sqrt(var(X)+eps)
		caffe_gpu_div(count_, top[0]->gpu_data(), Expand_Sigma_.gpu_data(), top[0]->mutable_gpu_data());
		// store for backpropogate
		caffe_copy(count_, top[0]->gpu_data(), Top_Buffer_.mutable_gpu_data());

	}

	template <typename Dtype>
	void InstanceNormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

		const Dtype* bottom_data = bottom[0]->gpu_data();
		const Dtype* top_diff = top[0]->gpu_diff();
		const Dtype* bottom_diff = bottom[0]->gpu_diff();
		Dtype* mutable_bottom_diff = bottom[0]->mutable_gpu_diff();
		const Dtype* top_data = Top_Buffer_.gpu_data();
		const Dtype* weight_sum = SumMatrix_.gpu_data();
		// y = (x - mu(x)) / var(x)
		// dE(Y)/dX =
		//   (dE/dY - mean(dE/dY) - mean(dE/dY \cdot Y) \cdot Y)
		//     ./ sqrt(var(X) + eps)

		// ----- sum(dE/dY \cdot Y) -----
		caffe_gpu_mul(count_, top_data, top_diff, mutable_bottom_diff);
		// sum 
		caffe_gpu_gemm(CblasNoTrans, CblasTrans, 1, ch_*num_, imgSize_, (Dtype)1, weight_sum, bottom_diff, (Dtype)0, Mu_.mutable_gpu_data());
		// extend
		caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, ch_*num_, imgSize_, 1, (Dtype)(1), Mu_.gpu_data(), weight_sum, (Dtype)0, mutable_bottom_diff);

		// ----- sum(dE/dY \cdot Y) \cdot Y -----
		caffe_gpu_mul(count_, top_data, bottom_diff, mutable_bottom_diff);

		// ----- sum(dE/dY) + sum(dE/dY \cdot Y) \cdot Y -----
		// sum 
		caffe_gpu_gemm(CblasNoTrans, CblasTrans, 1, ch_*num_, imgSize_, (Dtype)1, weight_sum, top_diff, (Dtype)0, Mu_.mutable_gpu_data());
		// extend
		caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, ch_*num_, imgSize_, 1, (Dtype)(1), Mu_.gpu_data(), weight_sum, (Dtype)1, mutable_bottom_diff);

		// ----- dE/dY - mean(dE/dY)-mean(dE/dY \cdot Y) \cdot Y -----
		caffe_gpu_axpby(count_, Dtype(1), top_diff, Dtype(-1. / imgSize_), mutable_bottom_diff);

		// ----- {dE/dY - mean(dE/dY)-mean(dE/dY \cdot Y) \cdot Y} / sqrt(var(X) + eps) -----
		caffe_gpu_div(count_, bottom_diff, Expand_Sigma_.gpu_data(), mutable_bottom_diff);


	}

	INSTANTIATE_LAYER_GPU_FUNCS(InstanceNormLayer);

}  // namespace caffe
